#include <iostream>
using namespace std;

#include "matmul_cuda.h"

/* If stmt evaluates to false; error out */
#define ERR_RET(stmt) if (stmt) { cerr<<"Error - "<<__FILE__<<" "<<__LINE__<<endl; return; }

/* Host setup for GPU execution done here */
void run_matmul_cuda(const float* A, const float* B, float* C, int mat_dim) {

  hipError_t err;

  float* devA = NULL;
  float* devB = NULL;
  float* devC = NULL;

  size_t data_bytes = mat_dim * mat_dim * sizeof(float);

  /* Allocate cuda device memory */
  err = hipMalloc(&devA, data_bytes);
  ERR_RET(err != hipSuccess)
  err = hipMalloc(&devB, data_bytes);
  ERR_RET(err != hipSuccess)
  err = hipMalloc(&devC, data_bytes);
  ERR_RET(err != hipSuccess)


  /* Copy Inputs from CPU to GPU */
  err = hipMemcpy(devA, A, data_bytes, hipMemcpyHostToDevice);
  ERR_RET(err != hipSuccess);
  err = hipMemcpy(devB, B, data_bytes, hipMemcpyHostToDevice);
  ERR_RET(err != hipSuccess);


  /* Copy Results from GPU to CPU */
  err = hipMemcpy(C, devC, data_bytes, hipMemcpyDeviceToHost);
  ERR_RET(err != hipSuccess);

  hipFree(devA);
  hipFree(devB);
  hipFree(devC);

  return;
}
